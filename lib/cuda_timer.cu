#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <thrust/host_vector.h>
#include "MemTimeTester.h"
#include "obs_calc_fast.cuh"
extern "C" {
#include "random.h"
}
/**
For my GPU, the maximum threads per multiprocessor is 2048, max per block is 1024, and the max dimensions are (1024, 1024, 64)
Maximum shared memory per block is 49152 bytes, L2 Cache is 2097152 bytes, 15 multiprocessors, 128 CUDA cores/MP
Cuda capability: 6.1, CUDA driver version 8.0
**/

double calc_action_slow(thrust::host_vector<double>& lat, thrust::host_vector<double>& interactions, int Lx, int Ly) {
	//essentially, the time-averaged energy of a given set of quantum fluctuations.  Calculate the action then divide by beta
	double S = 0;
	double s1;
	for (int i = 0; i < Lx; ++i) {
		for (int j = 0; j < Ly; ++j) {
			s1 = lat[i*Ly + j];
			for (int m = 0; m < Lx; ++m) {
				for (int n = 0; n < Ly; ++n) {
					S += s1 * lat[m*Ly + n] * interactions[((m - i + Lx) % Lx)*Ly + ((n - j + Ly) % Ly)];
				}
			}
		}
	}

	return 0.5*S;
}

thrust::host_vector<double> rand_vector(int length) {
	thrust::host_vector<double> result(length);
	for (int i = 0; i < length; ++i) {
		result[i] = drand1_();
	}
	return result;
}

thrust::host_vector<double> transpose(thrust::host_vector<double> s, int Lx, int Ly) {
	//take a vector indexed like s(x, y) = s[x*Ly + y] and make it s(x,y) = s[y*Lx + x]
	thrust::host_vector<double> new_s(Lx*Ly);
	for (int x = 0; x < Lx; ++x) {
		for (int y = 0; y < Ly; ++y) {
			new_s[y*Lx + x] = s[x*Ly + y];
		}
	}
	return new_s;
}

int main() {
	std::cout << "Timing different versions of fast calc and checking for accuracy of results\n";
	MemTimeTester timer;
	int seed = 1892347;
	rand_init_(&seed);
	double fast_action;
	double slow_action;
	
	/*Test fast action calculations
	*	1. Test accuracy for small calculations
	*	2. Test speed for some different versions
	*/

	//1. Test accuracy
	thrust::host_vector<double> acc_state = rand_vector(128);//just do something simple like lx = 8, ly = 16
	thrust::host_vector<double>& acc_state_ref = acc_state;
	thrust::host_vector<double> acc_int = rand_vector(128);//same dimensions
	thrust::host_vector<double>& acc_int_ref = acc_int;
	thrust::host_vector<double> acc_corr(128, 0.0);
	thrust::host_vector<double>& acc_corr_ref = acc_corr;
	slow_action = calc_action_slow(acc_state_ref, acc_int_ref, 8, 16);
	fast_action = cufft_calc_action(acc_state_ref, acc_int_ref, acc_corr_ref, 8, 16);
	if (slow_action == fast_action) {
		std::cout << "Action accuracy test passed!\n\n";
	}
	else {
		std::cout << "Action accuracy not quite right! Slow action: " 
			<< slow_action << ", fast action: " << fast_action << ", abs difference: " << abs(fast_action - slow_action) << "\n\n";
	}

	//2. Test timing
	thrust::host_vector<double> one_state(32 * 1024, 1.0);
	thrust::host_vector<double> one_state_2(32 * 1024, 1.0);
	thrust::host_vector<double> corr(32 * 1024, 1.0);
	thrust::host_vector<double>& corr_ref = corr;
	thrust::host_vector<double>& one_int_ref = one_state_2;
	thrust::host_vector<double>& one_state_ref = one_state;
	int Lx = 32, Ly = 1024;
	//no preallocation
	timer.flag_start_time("cufft total no prealloc");
	for (int i = 0; i < 10000; ++i) {
		fast_action = cufft_calc_action_timer(one_state_ref, one_int_ref, corr_ref, Lx, Ly, &timer);
		hipDeviceSynchronize();
		timer.flag_end_time("thrust inner product");
	}
	timer.flag_end_time("cufft total no prealloc");
	if (fast_action == 0.5 * 32 * 32 * 1024 * 1024) {
		std::cout << "Fast no prealloc action correct!\n";
	}
	else {
		std::cout << "Fast no prealloc action: " << fast_action << ", should be: " << 0.5 * 32 * 32 * 1024 * 1024 << "\n";
	}



	//with preallocation
	timer.flag_start_time("cufft total with prealloc");
	hipfftHandle forward_plan, backward_plan;
	hipError_t cuda_status;
	hipfftDoubleReal *state_rs;//real space state
	hipfftDoubleComplex *state_ft;//fourier space state
	int n[2] = { Lx, Ly };
	//allocate real state
	cuda_status = hipMalloc((void**)&state_rs, sizeof(hipfftDoubleReal)*Lx*Ly);
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "Cuda error: Failed to allocate. Code: %s\n", hipGetErrorName(cuda_status));
		show_memory();
		return 0;
	}

	//allocate transform state
	cuda_status = hipMalloc((void**)&state_ft, sizeof(hipfftDoubleComplex)*Lx*(Ly / 2 + 1));
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "Cuda error: Failed to allocate. Code: %s\n", hipGetErrorName(cuda_status));
		show_memory();
		return 0;
	}
	hipDeviceSynchronize();
	timer.flag_end_time("cufft allocation");

	timer.flag_start_time("cufft prealloc plan");
	//Create 2D R2C FFT plan
	if (hipfftPlanMany(&forward_plan, 2, n, NULL, 1, 0, NULL, 1, 0, HIPFFT_D2Z, 1) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT Error: Unable to create plan\n");
		return 0;
	}
	hipDeviceSynchronize();
	timer.flag_end_time("cufft prealloc plan");


	timer.flag_start_time("cufft prealloc plan");
	//Inverse fourier transform plan
	if (hipfftPlanMany(&backward_plan, 2, n, NULL, 1, 0, NULL, 1, 0, HIPFFT_Z2D, 1) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT Error: Unable to create plan\n");
		return 0;
	}
	hipDeviceSynchronize();
	timer.flag_end_time("cufft prealloc plan");

	for (int i = 0; i < 10000; ++i) {
		fast_action = cufft_calc_action_timer_prealloc(one_state_ref, one_int_ref, corr_ref, Lx, Ly, &timer,
			forward_plan,  backward_plan,  cuda_status,  state_rs,  state_ft);
	}

	timer.flag_start_time("cufft prealloc deallocate");
	//Deallocate memory
	hipfftDestroy(forward_plan);
	hipfftDestroy(backward_plan);
	hipFree(state_rs);
	hipFree(state_ft);
	hipDeviceSynchronize();
	timer.flag_end_time("cufft prealloc deallocate");
	timer.flag_end_time("cufft total with prealloc");

	if (fast_action == 0.5 * 32 * 32 * 1024 * 1024) {
		std::cout << "Fast prealloc action correct!\n";
	}
	else {
		std::cout << "Fast prealloc action: " << fast_action << ", should be: " << 0.5 * 32 * 32 * 1024 * 1024 << "\n";
	}

	timer.print_timers();

	return 0;
}