#include "MemTimeTester.h"
#include "hip/hip_runtime.h"
#include ""
#include "obs_calc_fast.cuh"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/count.h>
#include <thrust/inner_product.h>
#include <thrust/functional.h>
#include <thrust/fill.h>
#include <string>
#include <sstream>


#include <stdio.h>


int main(int argc, char* argv[])
{
	//first input must be the number of states in the dump directory
	std::stringstream input(argv[1]);
	int num_states;
	input >> num_states;

	MemTimeTester timer;
	std::ifstream file;
	std::string line;
	int Lx, Ly;


	//check average energy, correlation function
	std::cout << "Calculating average of " << num_states << " runs energy and correlation\n\n";
	char filename[300];

	thrust::host_vector<int> state;
	thrust::host_vector<int>& state_ref = state;
	thrust::host_vector<double> corr;
	thrust::host_vector<double>& corr_ref = corr;
	thrust::device_vector<double> corr_temp = corr;
	thrust::device_vector<double> corr_total;
	for (int run = 0; run < num_states; ++run) {
		sprintf(filename, "./dump/state%d.csv", run);
		file.open(filename);
		if (file.is_open()) {
			file >> Lx >> Ly;
			if(run == 0){
				state.resize(Lx*Ly);
			}
			for (int i = 0; i < Lx; ++i) {
				for (int j = 0; j < Ly; ++j) {
					file >> state[i*Ly + j];
				}
			}
		}
		else {
			std::cout << "Error: input file not opened\n";
		}
		file.close();
		if (run == 0){
			corr.resize(Lx*Ly);
			corr_temp.resize(Lx*Ly);
			corr_total.resize(Lx*Ly);
			thrust::fill(corr_total.begin(), corr_total.end(), 0.0);
		}
		timer.flag_start_time("double site correlation measurement");
		if(Lx == 1){
			calc_corr_fast_1site(corr_ref, state_ref, Ly);
		}
		else {
			calc_corr_fast_2site(corr_ref, state_ref, Ly);
		}
		//corr_temp = corr_ref;
		thrust::copy(corr_ref.begin(), corr_ref.end(), corr_temp.begin());
		thrust::transform(corr_temp.begin(), corr_temp.end(), corr_total.begin(), corr_total.begin(), thrust::plus<double>());
		timer.flag_end_time("double site correlation measurement");
	}
	thrust::constant_iterator<double> factor(1.0 / num_states);
	thrust::transform(corr_total.begin(), corr_total.end(), factor, corr_temp.begin(), thrust::multiplies<double>());
	//corr = corr_temp;
	thrust::copy(corr_temp.begin(), corr_temp.end(), corr.begin());
/*
	std::cout << "Double Site Correlation function:\n";
	for (int i = 0; i < corr.size(); ++i) {
		std::cout << corr[i] << ",";
	}
	std::cout << "\n";
*/
	std::ofstream outfile;
	outfile.open("corr.csv");
	for (int i = 0; i < Lx; ++i) {
		for (int j = 0 ; j < Ly - 1; ++j){
               		outfile  << corr[i*Ly + j] << ",";
		}
		outfile << corr[i * Ly + Ly - 1] << "\n";
        }
	outfile.close();

	timer.print_timers();

    return 0;
}
