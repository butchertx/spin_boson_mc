#include "hip/hip_runtime.h"
#include "obs_calc_fast.cuh"
#include <sstream>


#include <iterator>
#include <iostream>

template <typename InputIterator1,
	typename InputIterator2,
	typename OutputIterator>
	OutputIterator expand(InputIterator1 first1,
		InputIterator1 last1,
		InputIterator2 first2,
		OutputIterator output)
{
	typedef typename thrust::iterator_difference<InputIterator1>::type difference_type;

	difference_type input_size = thrust::distance(first1, last1);
	difference_type output_size = thrust::reduce(first1, last1);

	// scan the counts to obtain output offsets for each input element
	thrust::device_vector<difference_type> output_offsets(input_size, 0);
	thrust::exclusive_scan(first1, last1, output_offsets.begin());

	// scatter the nonzero counts into their corresponding output positions
	thrust::device_vector<difference_type> output_indices(output_size, 0);
	thrust::scatter_if
	(thrust::counting_iterator<difference_type>(0),
		thrust::counting_iterator<difference_type>(input_size),
		output_offsets.begin(),
		first1,
		output_indices.begin());

	// compute max-scan over the output indices, filling in the holes
	thrust::inclusive_scan
	(output_indices.begin(),
		output_indices.end(),
		output_indices.begin(),
		thrust::maximum<difference_type>());

	// gather input values according to index array (output = first2[output_indices])
	OutputIterator output_end = output; thrust::advance(output_end, output_size);
	thrust::gather(output_indices.begin(),
		output_indices.end(),
		first2,
		output);

	// return output + output_size
	thrust::advance(output, output_size);
	return output;
}


void calc_corr_fast_1site(thrust::host_vector<double>& corr, thrust::host_vector<int>& state, int Ly) {

	//begin calculation
	//copy to device
	thrust::device_vector<int> d_state = state;

	//calculate correlation
	//only works for Lx = 1 right now
	for (int i = 0; i < Ly; ++i) {
		corr[i] = 1.0 / (Ly) * (thrust::inner_product(d_state.begin(), d_state.end() - i, d_state.begin() + i, 0) + thrust::inner_product(d_state.begin(), d_state.begin() + i, d_state.end() - i, 0));
	}

}

void calc_corr_fast_2site(thrust::host_vector<double>& corr, thrust::host_vector<int>& state, int Ly) {
	thrust::device_vector<int> d_state = state;

	for (int i = 0; i < Ly; ++i) {
		corr[i] = 1.0 / (2 * Ly) * (thrust::inner_product(d_state.begin(), d_state.begin() + Ly - i, d_state.begin() + i, 0)
						+ thrust::inner_product(d_state.begin(), d_state.begin() + i, d_state.begin() + Ly - i, 0)
						+ thrust::inner_product(d_state.begin() + Ly, d_state.end() - i, d_state.begin() + Ly + i, 0)
						+ thrust::inner_product(d_state.begin() + Ly, d_state.begin() + Ly + i, d_state.end() - i, 0));
		corr[i + Ly] = 1.0 / (2 * Ly) * (thrust::inner_product(d_state.begin(), d_state.begin() + Ly - i, d_state.begin() + Ly + i, 0) 
						+ thrust::inner_product(d_state.begin(), d_state.begin() + i, d_state.end() - i, 0)
						+ thrust::inner_product(d_state.begin() + Ly, d_state.end() - i, d_state.begin() + i, 0) 
						+ thrust::inner_product(d_state.begin() + Ly, d_state.begin() + Ly + i, d_state.begin() + Ly - i, 0));
	}
}

double calc_action_fast(thrust::host_vector<double>& corr, thrust::host_vector<double>& interactions) {
	//interactions matrix at (i) should give interaction between sites x1 - x2 = i and interactions(0) = 0
	//this is actually the inner product of the correlation function with the interactions matrix
	return 0.5*thrust::inner_product(corr.begin(), corr.end(), interactions.begin(), 0.0);
}

__global__ void calc_shift_state(double* state, double* shift_state, int i, int j){
	//i and j are the "starting point" (0,0) of the new lattice state, held in shift_state
	//the lattice is shifted using pbc's to start at the new value of i,j
	//assume it is launched with gridDim.x = Lx and blockDim.x = Ly/32, blockDim.y = 32
	//position in shift_state is determined by thread variables, position in original state is relative to i and j
	//int Lx = gridDim.x, Ly = blockDim.x;
	//int shift_ind = blockIdx.x*Ly + threadIdx.x;//position in shift_state
	//int state_ind_x = (i + blockIdx.x) % Lx;//x coordinate of (i,j) + (m,n)
	//int state_ind_y = (j + threadIdx.x) % Ly;//y coordinate of (i,j) + (m,n)
	//int state_ind = state_ind_x*Ly + state_ind_y;
	//shift_state[shift_ind] = state[state_ind];
	//the above works when grids and blocks are 1D

	int Lx = gridDim.x, Ly = blockDim.x*blockDim.y*blockDim.z;
	int shift_ind = blockIdx.x*Ly + threadIdx.z*blockDim.y*blockDim.x + threadIdx.y*blockDim.x + threadIdx.x;//position in shift_state
	int state_ind_x = (i + blockIdx.x) % Lx;//x coordinate of (i,j) + (m,n)
	int state_ind_y = (j + threadIdx.z*blockDim.y*blockDim.x + threadIdx.y*blockDim.x + threadIdx.x) % Ly;//y coordinate of (i,j) + (m,n)
	int state_ind = state_ind_x*Ly + state_ind_y;
	shift_state[shift_ind] = state[state_ind];
}

double thrust_calc_action_general(thrust::host_vector<double>& state, thrust::host_vector<double>& interactions, int Lx, int Ly, dim3 threads){
	//
	//	Algorithm outline
	//	Def: L = state, A = interactions, Q_ij = sum_(m,n) L_(i + m, j + n) * A_(m,n), action = S = sum_(i,j) L_ij Q_ij
	//	2 steps: calculate Q_ij, then do L.Q (dot product)
	//	1. Q_ij calculation
	//
	if(state.size() != interactions.size() || state.size() != Lx*Ly){
		std::cout << "Error: state/interactions size mismatch\n";
		return 0;
	}
	thrust::device_vector<double> qij(Lx*Ly);
	thrust::device_vector<double> d_state = state;
	thrust::device_vector<double> shift_state = state;
	thrust::device_vector<double> d_int = interactions;
	for(int i = 0; i < Lx; ++i){
		for (int j = 0; j < Ly; ++j){
			calc_shift_state<<<Lx , threads>>>(thrust::raw_pointer_cast(&d_state[0]), thrust::raw_pointer_cast(&shift_state[0]), i, j);
			qij[i*Ly + j] = thrust::inner_product(shift_state.begin(), shift_state.end(), d_int.begin(), 0.0);
		}
	}
	return 0.5*thrust::inner_product(d_state.begin(), d_state.end(), qij.begin(), 0.0);
}

__global__ void elementwise_product_cmplx(hipfftDoubleComplex *source) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	source[i].x = source[i].x * source[i].x + source[i].y *source[i].y;
	source[i].y = 0;
}

double cufft_calc_action(thrust::host_vector<double>& state, thrust::host_vector<double>& interactions, thrust::host_vector<double>& corr, int Lx, int Ly) {
	//	Algorithm: Calculate the correlation function and then take the inner product with the interactions
	//	Correlation Calculation: correlation function is the inverse FT of |S_pr . S_pr|, calculated elementwise, where S_pr is the FT of the state vector

	hipfftHandle forward_plan, backward_plan;
	hipError_t cuda_status;

	hipfftDoubleReal *state_rs;//real space state

	hipfftDoubleComplex *state_ft;//fourier space state

	int n[2] = { Lx, Ly };

	//allocate real state
	cuda_status = hipMalloc((void**)&state_rs, sizeof(hipfftDoubleReal)*Lx*Ly);
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "Cuda error: Failed to allocate. Code: %s\n", hipGetErrorName(cuda_status));
		show_memory();
		return 0;
	}

	//allocate transform state
	cuda_status = hipMalloc((void**)&state_ft, sizeof(hipfftDoubleComplex)*Lx*(Ly/2 + 1));
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "Cuda error: Failed to allocate. Code: %s\n", hipGetErrorName(cuda_status));
		show_memory();
		return 0;
	}

	//copy vals for real state
	if (hipMemcpy(state_rs, thrust::raw_pointer_cast(&state[0]), sizeof(hipfftDoubleReal)*Lx*Ly, hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "Cuda error: Failed to copy\n");
		return 0;
	}

	//Create 2D R2C FFT plan
	if (hipfftPlanMany(&forward_plan, 2, n, NULL, 1, 0, NULL, 1, 0, HIPFFT_D2Z, 1) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT Error: Unable to create plan\n");
		return 0;
	}

	//Transform state
	if (hipfftExecD2Z(forward_plan, state_rs, state_ft) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT Error: Transform did not work\n");
		return 0;
	}

	//Multiply state_ft with itself.
	if (Ly / 2 >= 1024) {
		elementwise_product_cmplx<<<(Ly/2 + 1),Lx>>>(state_ft);
	}
	else{
		elementwise_product_cmplx<<<Lx, (Ly/2 + 1)>>>(state_ft);
	}

	//Inverse fourier transform plan
	if (hipfftPlanMany(&backward_plan, 2, n, NULL, 1, 0, NULL, 1, 0, HIPFFT_Z2D, 1) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT Error: Unable to create plan\n");
		return 0;
	}

	//Find correlation by taking the inverse fourier transform of state_ft
	if (hipfftExecZ2D(backward_plan, state_ft, state_rs) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT Error: Transform did not work\n");
		return 0;
	}

	//copy correlation back to a host vector
	if (hipMemcpy(thrust::raw_pointer_cast(&corr[0]), state_rs, sizeof(hipfftDoubleReal)*Lx*Ly, hipMemcpyDeviceToHost) != hipSuccess) {
		fprintf(stderr, "Cuda error: Failed to copy\n");
		return 0;
	}

	//Deallocate memory
	hipfftDestroy(forward_plan);
	hipfftDestroy(backward_plan);
	hipFree(state_rs);
	hipFree(state_ft);

	return 0.5*thrust::inner_product(interactions.begin(), interactions.end(), corr.begin(), 0.0) / ((double)Lx*Ly);
}

double cufft_calc_action_timer(thrust::host_vector<double>& state, thrust::host_vector<double>& interactions, 
		thrust::host_vector<double>& corr, int Lx, int Ly, MemTimeTester * timer){
		//hipfftHandle* forward_plan, hipfftHandle* backward_plan, hipError_t cuda_status, hipfftDoubleReal* state_rs, hipfftDoubleComplex* state_ft) {
	//	Algorithm: Calculate the correlation function and then take the inner product with the interactions
	//	Correlation Calculation: correlation function is the inverse FT of |S_pr . S_pr|, calculated elementwise, where S_pr is the FT of the state vector

	int n[2] = { Lx, Ly };

	timer->flag_start_time("cufft allocation");
	hipfftHandle forward_plan, backward_plan;
	hipError_t cuda_status;

	hipfftDoubleReal *state_rs;//real space state

	hipfftDoubleComplex *state_ft;//fourier space state



	//allocate real state
	cuda_status = hipMalloc((void**)&state_rs, sizeof(hipfftDoubleReal)*Lx*Ly);
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "Cuda error: Failed to allocate. Code: %s\n", hipGetErrorName(cuda_status));
		show_memory();
		return 0;
	}

	//allocate transform state
	cuda_status = hipMalloc((void**)&state_ft, sizeof(hipfftDoubleComplex)*Lx*(Ly / 2 + 1));
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "Cuda error: Failed to allocate. Code: %s\n", hipGetErrorName(cuda_status));
		show_memory();
		return 0;
	}
	hipDeviceSynchronize();
	timer->flag_end_time("cufft allocation");
	timer->flag_start_time("cufft copy");
	//copy vals for real state
	if (hipMemcpy(state_rs, thrust::raw_pointer_cast(&state[0]), sizeof(hipfftDoubleReal)*Lx*Ly, hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "Cuda error: Failed to copy\n");
		return 0;
	}
	hipDeviceSynchronize();
	timer->flag_end_time("cufft copy");
	timer->flag_start_time("cufft plan");
	//Create 2D R2C FFT plan
	if (hipfftPlanMany(&forward_plan, 2, n, NULL, 1, 0, NULL, 1, 0, HIPFFT_D2Z, 1) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT Error: Unable to create plan\n");
		return 0;
	}
	hipDeviceSynchronize();
	timer->flag_end_time("cufft plan");
	timer->flag_start_time("cufft exec");
	//Transform state
	if (hipfftExecD2Z(forward_plan, state_rs, state_ft) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT Error: Transform did not work\n");
		return 0;
	}
	hipDeviceSynchronize();
	timer->flag_end_time("cufft exec");
	timer->flag_start_time("cufft norm");
	//Multiply state_ft with itself.  If Ly/2 >= 1024, this needs to be modified
	if (Ly / 2 >= 1024) { std::cout << "Error: need to modify block structure to make correlation calculation correct\n"; return 0; }
	elementwise_product_cmplx << <Lx, (Ly / 2 + 1) >> >(state_ft);
	hipDeviceSynchronize();
	timer->flag_end_time("cufft norm");
	timer->flag_start_time("cufft plan");
	//Inverse fourier transform plan
	if (hipfftPlanMany(&backward_plan, 2, n, NULL, 1, 0, NULL, 1, 0, HIPFFT_Z2D, 1) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT Error: Unable to create plan\n");
		return 0;
	}
	hipDeviceSynchronize();
	timer->flag_end_time("cufft plan");
	timer->flag_start_time("cufft exec");
	//Find correlation by taking the inverse fourier transform of state_ft
	if (hipfftExecZ2D(backward_plan, state_ft, state_rs) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT Error: Transform did not work\n");
		return 0;
	}
	hipDeviceSynchronize();
	timer->flag_end_time("cufft exec");
	timer->flag_start_time("cufft copy");
	//copy correlation back to a host vector
	if (hipMemcpy(thrust::raw_pointer_cast(&corr[0]), state_rs, sizeof(hipfftDoubleReal)*Lx*Ly, hipMemcpyDeviceToHost) != hipSuccess) {
		fprintf(stderr, "Cuda error: Failed to copy\n");
		return 0;
	}
	hipDeviceSynchronize();
	timer->flag_end_time("cufft copy");
	timer->flag_start_time("cufft deallocate");
	//Deallocate memory
	hipfftDestroy(forward_plan);
	hipfftDestroy(backward_plan);
	hipFree(state_rs);
	hipFree(state_ft);
	hipDeviceSynchronize();
	timer->flag_end_time("cufft deallocate");
	timer->flag_start_time("thrust inner product");
	return 0.5*thrust::inner_product(interactions.begin(), interactions.end(), corr.begin(), 0.0) / ((double)Lx*Ly);
}

double cufft_calc_action_timer_prealloc(thrust::host_vector<double>& state, thrust::host_vector<double>& interactions,
	thrust::host_vector<double>& corr, int Lx, int Ly, MemTimeTester * timer,
	hipfftHandle forward_plan, hipfftHandle backward_plan, hipError_t cuda_status, hipfftDoubleReal *state_rs, hipfftDoubleComplex *state_ft) {
	//hipfftHandle* forward_plan, hipfftHandle* backward_plan, hipError_t cuda_status, hipfftDoubleReal* state_rs, hipfftDoubleComplex* state_ft) {
	//	Algorithm: Calculate the correlation function and then take the inner product with the interactions
	//	Correlation Calculation: correlation function is the inverse FT of |S_pr . S_pr|, calculated elementwise, where S_pr is the FT of the state vector


	timer->flag_start_time("cufft prealloc copy");
	//copy vals for real state
	if (hipMemcpy(state_rs, thrust::raw_pointer_cast(&state[0]), sizeof(hipfftDoubleReal)*Lx*Ly, hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "Cuda error: Failed to copy\n");
		return 0;
	}
	hipDeviceSynchronize();
	timer->flag_end_time("cufft prealloc copy");
	timer->flag_start_time("cufft prealloc exec");
	//Transform state
	if (hipfftExecD2Z(forward_plan, state_rs, state_ft) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT Error: Transform did not work\n");
		return 0;
	}
	hipDeviceSynchronize();
	timer->flag_end_time("cufft prealloc exec");
	timer->flag_start_time("cufft prealloc norm");
	//Multiply state_ft with itself.  If Ly/2 >= 1024, this needs to be modified
	if (Ly / 2 >= 1024) { std::cout << "Error: need to modify block structure to make correlation calculation correct\n"; return 0; }
	elementwise_product_cmplx << <Lx, (Ly / 2 + 1) >> >(state_ft);
	hipDeviceSynchronize();
	timer->flag_end_time("cufft prealloc norm");

	timer->flag_start_time("cufft prealloc exec");
	//Find correlation by taking the inverse fourier transform of state_ft
	if (hipfftExecZ2D(backward_plan, state_ft, state_rs) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT Error: Transform did not work\n");
		return 0;
	}
	hipDeviceSynchronize();
	timer->flag_end_time("cufft prealloc exec");
	timer->flag_start_time("cufft prealloc copy");
	//copy correlation back to a host vector
	if (hipMemcpy(thrust::raw_pointer_cast(&corr[0]), state_rs, sizeof(hipfftDoubleReal)*Lx*Ly, hipMemcpyDeviceToHost) != hipSuccess) {
		fprintf(stderr, "Cuda error: Failed to copy\n");
		return 0;
	}
	hipDeviceSynchronize();
	timer->flag_end_time("cufft prealloc copy");

	timer->flag_start_time("thrust inner product");
	return 0.5*thrust::inner_product(interactions.begin(), interactions.end(), corr.begin(), 0.0) / ((double)Lx*Ly);
}

double cufft_calc_action_prealloc(thrust::host_vector<double>& state, thrust::host_vector<double>& interactions,
	thrust::host_vector<double>& corr, int Lx, int Ly, hipfftHandle forward_plan, hipfftHandle backward_plan, hipError_t cuda_status, hipfftDoubleReal *state_rs, hipfftDoubleComplex *state_ft) {
	//perform the cufft calc action function, but use a pre-allocated workspace
	//copy vals for real state
	if (hipMemcpy(state_rs, thrust::raw_pointer_cast(&state[0]), sizeof(hipfftDoubleReal)*Lx*Ly, hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "Cufft_calc_action_prealloc error: Failed to copy real space state to device\n");
		return 0;
	}

	//Transform state
	if (hipfftExecD2Z(forward_plan, state_rs, state_ft) != HIPFFT_SUCCESS) {
		fprintf(stderr, "Cufft_calc_action_prealloc Error: Transform did not work\n");
		return 0;
	}

	//Multiply state_ft with itself.
	if (Ly / 2 >= 1024) {
		elementwise_product_cmplx<<<(Ly/2 + 1),Lx>>>(state_ft);
	}
	else{
		elementwise_product_cmplx<<<Lx, (Ly/2 + 1)>>>(state_ft);
	}

	//Find correlation by taking the inverse fourier transform of state_ft
	if (hipfftExecZ2D(backward_plan, state_ft, state_rs) != HIPFFT_SUCCESS) {
		fprintf(stderr, "Cufft_calc_action_prealloc Error: Transform did not work\n");
		return 0;
	}

	//copy correlation back to a host vector
	if (hipMemcpy(thrust::raw_pointer_cast(&corr[0]), state_rs, sizeof(hipfftDoubleReal)*Lx*Ly, hipMemcpyDeviceToHost) != hipSuccess) {
		fprintf(stderr, "Cufft_calc_action_prealloc error: Failed to copy correlation back to host\n");
		return 0;
	}

	return 0.5*thrust::inner_product(interactions.begin(), interactions.end(), corr.begin(), 0.0) / ((double)Lx*Ly);
}


__global__ void point_action_shift(double *state, double *shift_state, int x, int y, int Lx, int Ly) {
	//only use for Lx small, Ly < 1024
	int x_prime = blockIdx.x, y_prime = threadIdx.x;//state indices
	int i = (x_prime - x + Lx)%Lx * Ly + (y_prime - y + Ly)%Ly;
	shift_state[i] = state[x_prime*Ly + y_prime];
}

double calc_point_action(thrust::host_vector<double>& state, thrust::host_vector<double>& interactions, int x, int y, int Lx, int Ly){
	thrust::device_vector<double> state_d = state;
	thrust::device_vector<double> int_d = interactions;
	thrust::device_vector<double> shift_state_d(state.size());
	point_action_shift<<<Lx, Ly>>>(thrust::raw_pointer_cast(&state_d[0]), thrust::raw_pointer_cast(&shift_state_d[0]), x, y, Lx, Ly);
	return thrust::inner_product(int_d.begin(), int_d.end(), shift_state_d.begin(), 0.0);
}

void show_memory() {

	// show memory usage of GPU

	size_t free_byte;

	size_t total_byte;

	hipError_t cuda_status;

	cuda_status = hipMemGetInfo(&free_byte, &total_byte);

	if (hipSuccess != cuda_status) {

		printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status));

		exit(1);

	}



	double free_db = (double)free_byte;

	double total_db = (double)total_byte;

	double used_db = total_db - free_db;

	printf("GPU memory usage: used = %f MB, free = %f MB, total = %f MB\n",

		used_db / 1024.0 / 1024.0, free_db / 1024.0 / 1024.0, total_db / 1024.0 / 1024.0);
}